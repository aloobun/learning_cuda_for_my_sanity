#include<iostream>
#include<hip/hip_runtime.h>
using namespace std;

/* a kernel is launched as a grid of threads, a grid is a 3d array and for simiplicity we are working with 1d, 
blockid is index of current block, threadid is index of current thread, and blockDim specifoes the no. of threads in each dim of a block.*/

/* idx here is unique identifier for a thread with the entire grid, we do it to access elementd in the global memory*/

__global__ void vectadd_kernel(float *A, float *B, float *C, int n) {
	int idx= blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < n) {
		C[idx] = A[idx] + B[idx];
	}
}

__global__ void vectnumadd_kernel(int a, int b, int *c) {
	*c = a + b;
}

void vectadd(float *A, float *B, float *C, int n){
	int size = n * sizeof(float);
	float *d_A, *d_B, *d_C; //these are gpu pointers

	hipMalloc((void**)&d_A, size); //allocate memory with cudamalloc
	hipMalloc((void**)&d_B, size);
	hipMalloc((void**)&d_C, size);

	//before kernel call we transfer variables to device
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

	//kernel call
	vectadd_kernel<<<ceil(n/256.0), 256>>>(d_A, d_B, d_C, n);

	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost); //gpu to host

	//free device memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}

void vectnumadd(int a, int b, int *c){
	int *d_num_c;
	hipMalloc((void**)&d_num_c, sizeof(int));
	vectnumadd_kernel<<<1,1>>>(a, b, d_num_c);
	hipMemcpy(c, d_num_c, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_num_c);
}

int main() {
    int n = 1000;
    size_t size = n * sizeof(float); 
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    for(int i = 0; i < n; i++) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i * 2);
    }

    //vector add
    vectadd(h_A, h_B, h_C, n);

    //first and last elements of the vector
    cout << "C[0] = " << h_C[0] << endl;
    cout << "C[" << n-1 << "] = " << h_C[n-1] << endl;

    free(h_A);
    free(h_B);
    free(h_C);

    int r;
    vectnumadd(2,7, &r);
    cout << r << endl;
    return 0;
}
